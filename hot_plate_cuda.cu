
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<math.h>
#include<stdlib.h>
#include<sys/time.h>


#define MAXROW 1024
#define MAXCOL 1024

double when()
{
	struct timeval tp;
	gettimeofday(&tp, NULL);
	return ((double) tp.tv_sec + (double) tp.tv_usec * 1e-6);
}
void initialize(float *oA, float *nA)
{
	int i,j;
	for(i=0; i<MAXROW; i++)
	{
		for(j=0;j<MAXCOL;j++)
		{
			if(i==0 || j==0 || j==(MAXCOL-1))
			{
				nA[i*MAXCOL + j] = 0.0;
				oA[i*MAXCOL + j] = 0.0;
			}
			else if (i==MAXROW-1)
			{
				nA[i*MAXCOL + j] = 100.0;
				oA[i*MAXCOL + j] = 100.0;
			}
			else if (i==400 && j<=330)
			{
				nA[i*MAXCOL + j] = 100.0;
				oA[i*MAXCOL + j] = 100.0;
			}
			else if (i==200 && j ==500)
			{
				nA[i*MAXCOL + j] = 100.0;
				oA[i*MAXCOL + j] = 100.0;
			}
			else
			{
				nA[i*MAXCOL + j] = 50.0;
				oA[i*MAXCOL + j] = 50.0;
			}
		}
	}
}

__global__ void calculate_new_values(float *nA, float *oA)
{
	if(blockIdx.x == 0 || threadIdx.x ==0 ||blockIdx.x == MAXCOL-1 || threadIdx.x == MAXCOL-1 
		|| (blockIdx.x==400 && threadIdx.x<=330) || (blockIdx.x==200 && threadIdx.x==500)  )
	{}
	else
	{
		nA[blockIdx.x * MAXCOL + threadIdx.x] 
			=(oA[(blockIdx.x+1) * MAXCOL + threadIdx.x] + 
			oA[(blockIdx.x-1) * MAXCOL + threadIdx.x] + 
			oA[blockIdx.x * MAXCOL + threadIdx.x+1] + 
			oA[blockIdx.x * MAXCOL + threadIdx.x-1] + 
			(4 * oA[blockIdx.x * MAXCOL + threadIdx.x]))/8.0;
	}
}

int main(void)
{
	double start_time = when();
	float *nA, *oA;
	float *d_nA, *d_oA;
	float *tmp;
	
	int iter=0;
	float convergence;
	int converged = 0;
	
	int size = MAXROW * MAXCOL * sizeof(float);

	nA = (float*)malloc(size);
	oA = (float*)malloc(size);

	hipError_t err = hipMalloc((void**)&d_nA,size);
	hipError_t err1 = hipMalloc((void**)&d_oA,size);
	
	initialize(oA, nA);

	hipMemcpy(d_oA, oA, size, hipMemcpyHostToDevice);
	hipMemcpy(d_nA, nA, size, hipMemcpyHostToDevice);
	
	while(!converged)
	{
		calculate_new_values<<<1024,1024>>>(d_nA , d_oA);
		hipMemcpy(nA , d_nA, size, hipMemcpyDeviceToHost);
	
		converged = 1;
		for(int i=1;i<MAXROW-1;i++)
		{
			for(int j=1;j<MAXCOL-1;j++)
			{

				if( (i==400 && j<=330) || (i==200 && j ==500))
				{
								//skip
				}
				else
				{
					convergence = nA[i*MAXCOL + j]- ((nA[(i+1)*MAXCOL + j] + nA[(i-1)*MAXCOL + j] 
						+ nA[i*MAXCOL + j+1] + nA[i*MAXCOL + j-1])/4.0 );

					if(fabs(convergence) > 0.1)
					{
						converged = 0;
						break;
					}
				}
			}
			if(converged == 0)
				break;
		}
		iter++;

		tmp = d_nA;
		d_nA = d_oA;
		d_oA = tmp;
	}
	printf("iter = %d and execution time = %f\n",iter, when() - start_time);
			
	hipMemcpy(nA, d_nA, size, hipMemcpyDeviceToHost);
	
	hipFree(d_nA);
	hipFree(d_oA);
	free(nA);
	free(oA);	
	
	return 1;
		
}
